#include "hip/hip_runtime.h"
#include<mine.cuh>

__global__ void mine(BYTE *in, int *found, uint32_t *nonce, int size, int difficulty, uint32_t bucket, uint32_t max_nonce) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;

  uint32_t test = uint32_t(id) * bucket;
  uint32_t end = id == TOTAL - 1
    ? max_nonce
    : uint32_t(id + 1) * bucket;

  while(test < end && *found != FOUND) {
    if(verify(id, in, test, size, difficulty, found)) {
      *nonce = test;
    }

    test++;
  }
}

extern "C" {
  uint32_t cmine(const char *str, int difficulty, uint32_t max_nonce) {
    // host
    BYTE *buff = (BYTE *) str;
    int size = strlen(str);
    int res = NOT_FOUND;
    uint32_t n;

    uint64_t bucket = (uint64_t(max_nonce) + uint32_t(TOTAL- 1)) / uint32_t(TOTAL);

    // device
    BYTE *in;
    int *found;
    uint32_t *nonce;

    hipMalloc((void **)&in, size);
    hipMalloc((void **)&found, sizeof(int));
    hipMalloc((void **)&nonce, sizeof(uint32_t));

    hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);
    hipMemcpy(found, &res, sizeof(int), hipMemcpyHostToDevice);

    pre_sha256();
    mine<<< BLOCKS, THREADS >>>(in, found, nonce, size, difficulty, uint32_t(bucket), max_nonce);

    hipDeviceSynchronize();

    hipMemcpy(&res, found, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&n, nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipFree(in);
    hipFree(found);
    hipFree(nonce);

    return n;
  }
}
