#include "hip/hip_runtime.h"
#include<mine.cuh>

__global__ void mine(BYTE *in, int *found, uint32_t *nonce, int size, int difficulty) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;

  uint32_t test = uint32_t(id) * BUCKET;
  uint32_t end = id == TOTAL - 1
    ? MAX_NONCE
    : uint32_t(id + 1) * BUCKET;

  while(test < end && *found != FOUND) {
    if(verify(id, in, test, size, difficulty, found)) {
      *nonce = test;
    }

    test++;
  }
}

extern "C" {
  uint32_t cmine(const char *str, int difficulty) {
    // host
    BYTE *buff = (BYTE *) str;
    int size = strlen(str);
    int res = NOT_FOUND;
    uint32_t n;

    // device
    BYTE *in;
    int *found;
    uint32_t *nonce;

    hipMalloc((void **)&in, size);
    hipMalloc((void **)&found, sizeof(int));
    hipMalloc((void **)&nonce, sizeof(uint32_t));

    hipMemcpy(in, buff, size * sizeof(BYTE), hipMemcpyHostToDevice);
    hipMemcpy(found, &res, sizeof(int), hipMemcpyHostToDevice);

    pre_sha256();
    mine<<< BLOCKS, THREADS >>>(in, found, nonce, size, difficulty);

    hipDeviceSynchronize();

    hipMemcpy(&res, found, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&n, nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipFree(in);
    hipFree(found);
    hipFree(nonce);

    return n;
  }
}
